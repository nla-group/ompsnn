/*
MIT License

Copyright (c) 2022 Stefan Güttel, Xinye Chen

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include <iostream>
#include <vector>
#include <cstring>
#include <cmath>
#include <random>
#include <algorithm>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>

typedef float FLOAT;
typedef double DOUBLE;

#define __DEBUG__

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t stat = call; \
    if (stat != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS error: " << stat << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

#if defined(__DEBUG__)
template <class T>
void print_data(T* data, int n, int d) {
    std::vector<T> h_data(n * d);
    CHECK_CUDA(hipMemcpy(h_data.data(), data, n * d * sizeof(T), hipMemcpyDeviceToHost));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < d; j++) {
            std::cout << std::setw(3) << std::setprecision(3) << h_data[i * d + j] << "   ";
        }
        std::cout << std::endl;
    }
}
#endif

// CUDA kernel for centering data (row-major)
__global__ void center_data_kernel(FLOAT* data, const FLOAT* mean, int n, int d) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < d) {
        data[i * d + j] -= mean[j];
    }
}

// CUDA kernel for computing squared norms (row-major)
__global__ void compute_norms_kernel(FLOAT* data, FLOAT* norms, int n, int d) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        FLOAT norm_sq = 0.0f;
        for (int j = 0; j < d; j++) {
            FLOAT val = data[i * d + j];
            norm_sq += val * val;
        }
        norms[i] = norm_sq;
    }
}

class SNN_FLOAT {
private:
    int n; // Number of samples
    int d; // Number of features
    FLOAT* d_data; // Device-centered data (n x d, row-major)
    FLOAT* d_sorted_proj; // Device projections
    int* d_indices; // Device indices
    FLOAT* d_norms; // Device squared norms
    std::vector<std::tuple<FLOAT, int, FLOAT>> sorted_proj_idx; // Host sorted data
    hipblasHandle_t cublas_handle;

    void compute_projections_and_norms(FLOAT* d_projections) {
        float alpha = 1.0f, beta = 0.0f;
        // For row-major d_data (n x d), use CUBLAS_OP_T to treat it as d x n column-major
        CHECK_CUBLAS(hipblasSgemv(cublas_handle, HIPBLAS_OP_T, d, n, &alpha, d_data, d,
                                 d_first_pc, 1, &beta, d_projections, 1));

        int threads_per_block = 256;
        int blocks = (n + threads_per_block - 1) / threads_per_block;
        compute_norms_kernel<<<blocks, threads_per_block>>>(d_data, d_norms, n, d);
        CHECK_CUDA(hipDeviceSynchronize());

        // Transfer to host and sort
        std::vector<FLOAT> h_projections(n);
        std::vector<int> h_indices(n);
        std::vector<FLOAT> h_norms(n);
        CHECK_CUDA(hipMemcpy(h_projections.data(), d_projections, n * sizeof(FLOAT), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_norms.data(), d_norms, n * sizeof(FLOAT), hipMemcpyDeviceToHost));
        for (int i = 0; i < n; i++) {
            sorted_proj_idx[i] = std::make_tuple(h_projections[i], i, h_norms[i]);
        }
        std::sort(sorted_proj_idx.begin(), sorted_proj_idx.end(),
                  [](const auto& a, const auto& b) { return std::get<0>(a) < std::get<0>(b); });

        for (int i = 0; i < n; i++) {
            h_projections[i] = std::get<0>(sorted_proj_idx[i]);
            h_indices[i] = std::get<1>(sorted_proj_idx[i]);
            h_norms[i] = std::get<2>(sorted_proj_idx[i]);
        }
        CHECK_CUDA(hipMemcpy(d_sorted_proj, h_projections.data(), n * sizeof(FLOAT), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_indices, h_indices.data(), n * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_norms, h_norms.data(), n * sizeof(FLOAT), hipMemcpyHostToDevice));
    }

public:
    FLOAT* d_mean; // Device feature means (d)
    FLOAT* d_first_pc; // Device first principal component (d x 1)

    SNN_FLOAT(FLOAT* input_data, int num_samples, int num_features)
        : n(num_samples), d(num_features), sorted_proj_idx(n) {
        CHECK_CUBLAS(hipblasCreate(&cublas_handle));

        CHECK_CUDA(hipMalloc(&d_data, n * d * sizeof(FLOAT)));
        CHECK_CUDA(hipMalloc(&d_mean, d * sizeof(FLOAT)));
        CHECK_CUDA(hipMalloc(&d_first_pc, d * sizeof(FLOAT)));
        CHECK_CUDA(hipMalloc(&d_sorted_proj, n * sizeof(FLOAT)));
        CHECK_CUDA(hipMalloc(&d_indices, n * sizeof(int)));
        CHECK_CUDA(hipMalloc(&d_norms, n * sizeof(FLOAT)));

        CHECK_CUDA(hipMemcpy(d_data, input_data, n * d * sizeof(FLOAT), hipMemcpyHostToDevice));
        compute_first_pc();
    }

    ~SNN_FLOAT() {
        CHECK_CUDA(hipFree(d_data));
        CHECK_CUDA(hipFree(d_mean));
        CHECK_CUDA(hipFree(d_first_pc));
        CHECK_CUDA(hipFree(d_sorted_proj));
        CHECK_CUDA(hipFree(d_indices));
        CHECK_CUDA(hipFree(d_norms));
        CHECK_CUBLAS(hipblasDestroy(cublas_handle));
    }

    std::vector<FLOAT> get_first_pc() const {
        std::vector<FLOAT> h_first_pc(d);
        CHECK_CUDA(hipMemcpy(h_first_pc.data(), d_first_pc, d * sizeof(FLOAT), hipMemcpyDeviceToHost));
        return h_first_pc;
    }

private:
    void compute_first_pc() {
        // Step 1: Compute mean and center data
        std::vector<FLOAT> h_mean(d, 0.0f);
        for (int j = 0; j < d; j++) {
            float sum;
            CHECK_CUBLAS(hipblasSasum(cublas_handle, n, d_data + j, d, &sum));
            h_mean[j] = sum / n;
        }
        CHECK_CUDA(hipMemcpy(d_mean, h_mean.data(), d * sizeof(FLOAT), hipMemcpyHostToDevice));

        dim3 threads(16, 16);
        dim3 blocks((n + threads.x - 1) / threads.x, (d + threads.y - 1) / threads.y);
        center_data_kernel<<<blocks, threads>>>(d_data, d_mean, n, d);
        CHECK_CUDA(hipDeviceSynchronize());

        // Step 2: Power iteration
        std::mt19937 gen(std::random_device{}());
        std::uniform_real_distribution<FLOAT> dis(-1.0f, 1.0f);
        std::vector<FLOAT> h_first_pc(d);
        for (int i = 0; i < d; i++) h_first_pc[i] = dis(gen);
        CHECK_CUDA(hipMemcpy(d_first_pc, h_first_pc.data(), d * sizeof(FLOAT), hipMemcpyHostToDevice));

        FLOAT* d_temp;
        CHECK_CUDA(hipMalloc(&d_temp, n * sizeof(FLOAT)));
        CHECK_CUDA(hipMemset(d_temp, 0, n * sizeof(FLOAT)));
        float alpha = 1.0f, beta = 0.0f;
        const int max_iter = 100;
        const FLOAT tol = 1e-6f;
        FLOAT norm_prev = 0.0f;

        CHECK_CUDA(hipDeviceSynchronize());

        for (int iter = 0; iter < max_iter; iter++) {
            // For row-major d_data (n x d), treat as d x n column-major with CUBLAS_OP_T
            CHECK_CUBLAS(hipblasSgemv(cublas_handle, HIPBLAS_OP_T, d, n, &alpha, d_data, d,
                                     d_first_pc, 1, &beta, d_temp, 1));
            // Second call: d_data^T (n x d row-major) as d x n column-major
            CHECK_CUBLAS(hipblasSgemv(cublas_handle, HIPBLAS_OP_N, d, n, &alpha, d_data, d,
                                     d_temp, 1, &beta, d_first_pc, 1));

            FLOAT norm;
            CHECK_CUBLAS(hipblasSnrm2(cublas_handle, d, d_first_pc, 1, &norm));
            if (norm < 1e-10f) {
                std::cerr << "Zero norm encountered\n";
                break;
            }
            float scale = alpha / norm;
            CHECK_CUBLAS(hipblasSscal(cublas_handle, d, &scale, d_first_pc, 1));

            if (iter > 0 && std::abs(norm - norm_prev) < tol * norm) {
                break;
            }
            norm_prev = norm;
        }

        FLOAT* d_projections;
        CHECK_CUDA(hipMalloc(&d_projections, n * sizeof(FLOAT)));
        compute_projections_and_norms(d_projections);
        CHECK_CUDA(hipFree(d_temp));
        CHECK_CUDA(hipFree(d_projections));
    }

public:
    std::vector<int> query_radius(const FLOAT* new_data, FLOAT R) const {
        FLOAT R_sq = R * R;

        FLOAT* d_centered;
        CHECK_CUDA(hipMalloc(&d_centered, d * sizeof(FLOAT)));
        std::vector<FLOAT> h_centered(d);
        for (int j = 0; j < d; j++) h_centered[j] = new_data[j];
        CHECK_CUDA(hipMemcpy(d_centered, h_centered.data(), d * sizeof(FLOAT), hipMemcpyHostToDevice));

        int threads_per_block = 256;
        int blocks = (d + threads_per_block - 1) / threads_per_block;
        center_data_kernel<<<blocks, threads_per_block>>>(d_centered, d_mean, 1, d);
        CHECK_CUDA(hipDeviceSynchronize());

        FLOAT q;
        CHECK_CUBLAS(hipblasSdot(cublas_handle, d, d_first_pc, 1, d_centered, 1, &q));
        FLOAT new_norm_sq;
        CHECK_CUBLAS(hipblasSdot(cublas_handle, d, d_centered, 1, d_centered, 1, &new_norm_sq));

        FLOAT* d_dot_products;
        CHECK_CUDA(hipMalloc(&d_dot_products, n * sizeof(FLOAT)));
        float alpha = 1.0f, beta = 0.0f;
        // d_data (n x d row-major) as d x n column-major with CUBLAS_OP_T
        CHECK_CUBLAS(hipblasSgemv(cublas_handle, HIPBLAS_OP_T, d, n, &alpha, d_data, d,
                                 d_centered, 1, &beta, d_dot_products, 1));

        std::vector<FLOAT> h_dot_products(n);
        CHECK_CUDA(hipMemcpy(h_dot_products.data(), d_dot_products, n * sizeof(FLOAT), hipMemcpyDeviceToHost));

        auto lower_it = std::lower_bound(sorted_proj_idx.begin(), sorted_proj_idx.end(),
                                         q - R,
                                         [](const auto& p, FLOAT val) { return std::get<0>(p) < val; });
        auto upper_it = std::upper_bound(sorted_proj_idx.begin(), sorted_proj_idx.end(),
                                         q + R,
                                         [](FLOAT val, const auto& p) { return val < std::get<0>(p); });

        std::vector<int> indices;
        indices.reserve(upper_it - lower_it);
        for (auto it = lower_it; it != upper_it; ++it) {
            int idx = std::get<1>(*it);
            FLOAT dot_xy = h_dot_products[idx];
            FLOAT norm_sq = std::get<2>(*it);
            FLOAT dist_sq = norm_sq + new_norm_sq - 2.0f * dot_xy;
            if (dist_sq <= R_sq) indices.push_back(idx);
        }

        CHECK_CUDA(hipFree(d_centered));
        CHECK_CUDA(hipFree(d_dot_products));
        return indices;
    }

    std::vector<std::vector<int>> query_radius_batch(const FLOAT* new_data, int m, FLOAT R) const {
        FLOAT R_sq = R * R;

        FLOAT* d_centered;
        CHECK_CUDA(hipMalloc(&d_centered, m * d * sizeof(FLOAT)));
        CHECK_CUDA(hipMemcpy(d_centered, new_data, m * d * sizeof(FLOAT), hipMemcpyHostToDevice));
        dim3 threads(16, 16);
        dim3 blocks((m + threads.x - 1) / threads.x, (d + threads.y - 1) / threads.y);
        center_data_kernel<<<blocks, threads>>>(d_centered, d_mean, m, d);
        CHECK_CUDA(hipDeviceSynchronize());

        FLOAT* d_q_values;
        CHECK_CUDA(hipMalloc(&d_q_values, m * sizeof(FLOAT)));
        float alpha = 1.0f, beta = 0.0f;
        // d_centered (m x d row-major) as d x m column-major with CUBLAS_OP_T
        CHECK_CUBLAS(hipblasSgemv(cublas_handle, HIPBLAS_OP_T, d, m, &alpha, d_centered, d,
                                 d_first_pc, 1, &beta, d_q_values, 1));

        FLOAT* d_new_norm_sq;
        CHECK_CUDA(hipMalloc(&d_new_norm_sq, m * sizeof(FLOAT)));
        int threads_per_block = 256;
        int norm_blocks = (m + threads_per_block - 1) / threads_per_block;
        compute_norms_kernel<<<norm_blocks, threads_per_block>>>(d_centered, d_new_norm_sq, m, d);
        CHECK_CUDA(hipDeviceSynchronize());

        FLOAT* d_dot_products;
        CHECK_CUDA(hipMalloc(&d_dot_products, n * m * sizeof(FLOAT)));
        // d_data (n x d row-major) as d x n, d_centered (m x d row-major) as d x m
        CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, d, &alpha,
                                 d_data, d, d_centered, d, &beta, d_dot_products, n));

        std::vector<FLOAT> h_q_values(m);
        std::vector<FLOAT> h_new_norm_sq(m);
        std::vector<FLOAT> h_dot_products(n * m);
        CHECK_CUDA(hipMemcpy(h_q_values.data(), d_q_values, m * sizeof(FLOAT), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_new_norm_sq.data(), d_new_norm_sq, m * sizeof(FLOAT), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_dot_products.data(), d_dot_products, n * m * sizeof(FLOAT), hipMemcpyDeviceToHost));

        std::vector<std::vector<int>> all_indices(m);
        for (int j = 0; j < m; j++) {
            FLOAT q = h_q_values[j];
            auto lower_it = std::lower_bound(sorted_proj_idx.begin(), sorted_proj_idx.end(),
                                             q - R,
                                             [](const auto& p, FLOAT val) { return std::get<0>(p) < val; });
            auto upper_it = std::upper_bound(sorted_proj_idx.begin(), sorted_proj_idx.end(),
                                             q + R,
                                             [](FLOAT val, const auto& p) { return val < std::get<0>(p); });

            std::vector<int>& indices = all_indices[j];
            indices.reserve(upper_it - lower_it);

            for (auto it = lower_it; it != upper_it; ++it) {
                int idx = std::get<1>(*it);
                FLOAT norm_sq = std::get<2>(*it);
                FLOAT dot_xy = h_dot_products[idx + j * n];
                FLOAT dist_sq = norm_sq + h_new_norm_sq[j] - 2.0f * dot_xy;
                if (dist_sq <= R_sq) indices.push_back(idx);
            }
        }

        CHECK_CUDA(hipFree(d_centered));
        CHECK_CUDA(hipFree(d_q_values));
        CHECK_CUDA(hipFree(d_new_norm_sq));
        CHECK_CUDA(hipFree(d_dot_products));
        return all_indices;
    }
};

#if defined(__DEBUG__)
int main() {
    int n = 7;
    int d = 3;
    std::vector<FLOAT> data = {
        1.2f, 2.0f, 3.0f,
        2.0f, 2.4f, 2.0f,
        2.0f, 1.0f, 2.0f,
        2.0f, 3.2f, 1.2f,
        2.0f, 3.1f, 2.0f,
        2.0f, 2.2f, 1.0f,
        2.0f, 2.1f, 1.0f
    };

    SNN_FLOAT snn_index(data.data(), n, d);

    std::cout << "mean:" << std::endl;
    print_data(snn_index.d_mean, 1, d);

    std::cout << "first principal:" << std::endl;
    print_data(snn_index.d_first_pc, 1, d);

    FLOAT R = 2.0f;

    std::cout << "Single query:" << std::endl;
    std::vector<FLOAT> new_data_unit = {2.3f, 3.2f, 1.0f};
    std::vector<int> indices = snn_index.query_radius(new_data_unit.data(), R);

    std::cout << "Found " << indices.size() << " indices within distance " << R << ":\n";
    std::cout << "Index: ";
    for (int i = 0; i < indices.size(); i++) {
        std::cout << indices[i] << " ";
    }
    std::cout << "\n";

    std::cout << "Multiple queries:" << std::endl;
    std::vector<FLOAT> new_data = {
        2.3f, 2.5f, 1.0f,
        1.5f, 2.5f, 2.5f,
        2.1f, 1.8f, 1.2f
    };
    int m = 3;
    std::vector<std::vector<int>> all_indices = snn_index.query_radius_batch(new_data.data(), m, R);

    for (int j = 0; j < m; j++) {
        std::cout << "Query " << j << " found " << all_indices[j].size() << " indices within distance " << R << ":\n";
        std::cout << "Index: ";
        for (int idx : all_indices[j]) {
            std::cout << idx << " ";
        }
        std::cout << "\n";
    }
    std::cout << "\n";

    return 0;
}
#endif